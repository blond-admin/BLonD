
#include <hip/hip_runtime.h>
#ifdef USEFLOAT
    typedef float real_t;
#else
    typedef double real_t;
#endif

extern "C"
__global__ void drift_simple(
                     real_t * __restrict__ beam_dt,
                     real_t * __restrict__ beam_dE,
                     const real_t T,
                     const real_t eta_zero,
                     const real_t beta,
                     const real_t energy,
                     const int n_macroparticles
                     )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    real_t coeff = T * eta_zero / (beta * beta * energy);
    for (int i=tid; i<n_macroparticles; i=i+blockDim.x*gridDim.x)
        beam_dt[i] +=  coeff * beam_dE[i];
}


extern "C"
__global__ void kick_single_harmonic(
    real_t  * __restrict__ beam_dt,
    real_t  * __restrict__ beam_dE,
    const real_t charge,
    const real_t voltage,
    const real_t omega_RF,
    const real_t phi_RF,
    const int n_macroparticles,
    const real_t acc_kick
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        beam_dE[i] += charge * voltage * sin(omega_RF*beam_dt[i] + phi_RF) + acc_kick;
    }
}

extern "C"
__global__ void kick_multi_harmonic(
    real_t  * __restrict__ beam_dt,
    real_t  * __restrict__ beam_dE,
    const int n_rf,
    const real_t charge,
    const real_t  * __restrict__ voltage,
    const real_t  * __restrict__ omega_RF,
    const real_t  * __restrict__ phi_RF,
    const int n_macroparticles,
    const real_t acc_kick
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    real_t my_beam_dt;
    real_t my_beam_dE;

    if (n_rf == 1) {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x)
            beam_dE[i] += charge * voltage[0] * sin(omega_RF[0]*beam_dt[i] + phi_RF[0]) + acc_kick;

    } else if (n_rf == 2) {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x){
            const real_t dE_sum = (
                charge * voltage[0] * sin(omega_RF[0]*beam_dt[i] + phi_RF[0])
              + charge * voltage[1] * sin(omega_RF[1]*beam_dt[i] + phi_RF[1])
              );
            beam_dE[i] += dE_sum + acc_kick;
        }

    } else if (n_rf == 3) {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x){
            const real_t dE_sum = (
                charge * voltage[0] * sin(omega_RF[0]*beam_dt[i] + phi_RF[0])
              + charge * voltage[1] * sin(omega_RF[1]*beam_dt[i] + phi_RF[1])
              + charge * voltage[2] * sin(omega_RF[2]*beam_dt[i] + phi_RF[2])
              );
            beam_dE[i] += dE_sum + acc_kick;
        }
    } else if (n_rf == 4) {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x){
            const real_t dE_sum = (
                charge * voltage[0] * sin(omega_RF[0]*beam_dt[i] + phi_RF[0])
              + charge * voltage[1] * sin(omega_RF[1]*beam_dt[i] + phi_RF[1])
              + charge * voltage[2] * sin(omega_RF[2]*beam_dt[i] + phi_RF[2])
              + charge * voltage[3] * sin(omega_RF[3]*beam_dt[i] + phi_RF[3])
              );
            beam_dE[i] += dE_sum + acc_kick;
        }
    } else {
        for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
            my_beam_dt = beam_dt[i];
            my_beam_dE = beam_dE[i];
            for (int j = 0; j < n_rf; j++) {
                my_beam_dE += charge * voltage[j] * sin(omega_RF[j]*my_beam_dt + phi_RF[j]);
            }
            beam_dE[i] = my_beam_dE + acc_kick;
        }
    }



}


extern "C"
__global__ void beam_phase(const real_t* __restrict__ hist_x,
                           const real_t* __restrict__ hist_y,
                           real_t* result,
                           real_t alpha,
                           real_t omega_rf,
                           real_t phi_rf,
                           real_t bin_size,
                           int n_bins)
{
    extern __shared__ real_t shared[];

    real_t* sin_partial = shared;
    real_t* cos_partial = shared + blockDim.x;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    real_t sin_val = 0.0;
    real_t cos_val = 0.0;

    if (i < n_bins) {
        real_t x = hist_x[i];
        real_t prof = hist_y[i];
        real_t phase = omega_rf * x + phi_rf;
        real_t base = exp(alpha * x) * prof;

        real_t coeff = ((i == 0) || (i == n_bins - 1)) ? 1.0 : 2.0;

        sin_val = coeff * base * sin(phase);
        cos_val = coeff * base * cos(phase);
    }

    sin_partial[threadIdx.x] = sin_val;
    cos_partial[threadIdx.x] = cos_val;

    __syncthreads();

    // Parallel reduction within block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sin_partial[threadIdx.x] += sin_partial[threadIdx.x + s];
            cos_partial[threadIdx.x] += cos_partial[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Only thread 0 adds to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&result[0], sin_partial[0]);
        atomicAdd(&result[1], cos_partial[0]);
    }
}



extern "C"
__global__ void hybrid_histogram(
                                 const real_t * __restrict__  input,
                                 real_t * __restrict__  output,
                                 const real_t cut_left,
                                 const real_t cut_right,
                                 const unsigned int n_slices,
                                 const unsigned int n_macroparticles,
                                 const int capacity
                                 )
{
    extern __shared__ int block_hist[];
    //reset shared memory
    for (int i = threadIdx.x; i < capacity; i += blockDim.x)
        block_hist[i] = 0;
    __syncthreads();
    int const tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    real_t const inv_bin_width = n_slices / (cut_right - cut_left);

    const int low_tbin = (n_slices / 2) - (capacity / 2);
    const int high_tbin = low_tbin + capacity;


    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        if (input[i] == cut_right){
            target_bin = n_slices - 1;
            if (target_bin >= low_tbin && target_bin < high_tbin)
                atomicAdd(&(block_hist[target_bin - low_tbin]), 1);
            else
                atomicAdd(&(output[target_bin]), 1);
            continue;
        }
        target_bin = floor((input[i] - cut_left) * inv_bin_width);
        if (target_bin < 0 || target_bin >= n_slices)
            continue;
        if (target_bin >= low_tbin && target_bin < high_tbin)
            atomicAdd(&(block_hist[target_bin - low_tbin]), 1);
        else
            atomicAdd(&(output[target_bin]), 1);

    }
    __syncthreads();
    for (int i = threadIdx.x; i < capacity; i += blockDim.x)
        atomicAdd(&output[low_tbin + i], (real_t) block_hist[i]);
}


extern "C"
__global__ void sm_histogram(const real_t * __restrict__  input,
                             real_t * __restrict__  output,
                             const real_t cut_left,
                             const real_t cut_right,
                             const unsigned int n_slices,
                             const unsigned int n_macroparticles)
{
    extern __shared__ int block_hist[];
    for (int i = threadIdx.x; i < n_slices; i += blockDim.x)
        block_hist[i] = 0;
    __syncthreads();
    int const tid = threadIdx.x + blockDim.x * blockIdx.x;
    int target_bin;
    real_t const inv_bin_width = n_slices / (cut_right - cut_left);
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        target_bin = floor((input[i] - cut_left) * inv_bin_width);

        if (input[i] == cut_right){
            target_bin = n_slices - 1;
            atomicAdd(&(block_hist[target_bin]), 1);
            continue;
        }

        if (target_bin < 0 || target_bin >= n_slices)
            continue;

        atomicAdd(&(block_hist[target_bin]), 1);
    }
    __syncthreads();
    for (int i = threadIdx.x; i < n_slices; i += blockDim.x)
        atomicAdd(&output[i], (real_t) block_hist[i]);
}




extern "C"
__global__ void lik_only_gm_copy(
    real_t * __restrict__ beam_dt,
    real_t * __restrict__ beam_dE,
    const real_t * __restrict__ voltage_array,
    const real_t * __restrict__ bin_centers,
    const real_t charge,
    const int n_slices,
    const int n_macroparticles,
    const real_t acc_kick,
    real_t * __restrict__ glob_vkick_factor
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    real_t const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);


    for (int i = tid; i < n_slices - 1; i += gridDim.x * blockDim.x) {
        glob_vkick_factor[2*i] = charge * (voltage_array[i + 1] - voltage_array[i])
                              * inv_bin_width;
        glob_vkick_factor[2*i+1] = (charge * voltage_array[i] - bin_centers[i] * glob_vkick_factor[2*i])
                         + acc_kick;
    }
}


extern "C"
__global__ void lik_only_gm_comp(
    real_t * __restrict__ beam_dt,
    real_t * __restrict__ beam_dE,
    const real_t * __restrict__ voltage_array,
    const real_t * __restrict__ bin_centers,
    const real_t charge,
    const int n_slices,
    const int n_macroparticles,
    const real_t acc_kick,
    real_t * __restrict__ glob_vkick_factor
)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    real_t const inv_bin_width = (n_slices - 1)
                                 / (bin_centers[n_slices - 1] - bin_centers[0]);
    int fbin;
    const real_t bin0 = bin_centers[0];
    for (int i = tid; i < n_macroparticles; i += blockDim.x * gridDim.x) {
        fbin = floor((beam_dt[i] - bin0) * inv_bin_width);
        if ((fbin < n_slices - 1) && (fbin >= 0))
            beam_dE[i] += beam_dt[i] * glob_vkick_factor[2*fbin] + glob_vkick_factor[2*fbin+1];
    }
}
